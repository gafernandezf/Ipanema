
#include <hip/hip_runtime.h>
__global__ void transform_f32(float *in, float *out, float *T, int N)
  {
    int el = threadIdx.x + blockDim.x * blockIdx.x;
    int i0 = el*N;
    int i, j;
    for (i = 0; i <N; i+=1){
        out[i0 + i] = 0;
        for (j = 0; j<N; j+=1) {
           out[i0 + i]  +=  T[j + i*N]  *in[i0 +j];
        }
    }
    
  }