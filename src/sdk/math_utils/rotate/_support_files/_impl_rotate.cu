
#include <hip/hip_runtime.h>
// Original code Copyright (C) Diego Martinez Santos
// Licensed under the GNU Affero General Public License v3.0
// Modifications Copyright (C) 2025 Gabriel Alejandro Fernandez Fernandez

__global__ void transform_f32(float *in, float *out, float *T, int N)
  {
    int el = threadIdx.x + blockDim.x * blockIdx.x;
    int i0 = el*N;
    int i, j;
    for (i = 0; i <N; i+=1){
        out[i0 + i] = 0;
        for (j = 0; j<N; j+=1) {
           out[i0 + i]  +=  T[j + i*N]  *in[i0 +j];
        }
    }
    
  }